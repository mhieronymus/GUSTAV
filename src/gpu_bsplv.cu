#include "gpu_bsplv.h"

/*
 * Allocate memory on the GPU and move over the splinetable.
 * We ommit stuff such as aux. Hence we don't copy the whole struct but we
 * have to bind the arrays to the struct.
 */
 GPU_BSPLV::GPU_BSPLV(splinetable * table)
{
    hipMalloc(&Table, sizeof(*Table));    

    // Allocate the arrays
    hipMalloc(&Order, table->ndim*sizeof(*Order));
    // Get the total number of knots
    index_t total_nknots = 0;
    for(index_t k=0; k<table->ndim; ++k) total_nknots += table->nknots[k];
    hipMalloc(&Knots, total_nknots*sizeof(*Knots));
    hipMalloc(&Nknots, table->ndim*sizeof(*Nknots));
    // Extends?
    // Periods?
    // Get the total number of coefficients
    index_t n_coeffs = 0;
    for(index_t c=0; c<table->ndim; ++c) n_coeffs += table->naxes[c];
    hipMalloc(&Coefficients, n_coeffs*sizeof(*Coefficients));
    hipMalloc(&Naxes, table->ndim*sizeof(*Naxes));
    hipMalloc(&Strides, table->ndim*sizeof(*Strides));

    // Copy the data
    hipMemcpy(Order, 
        table->order, table->ndim*sizeof(table->order[0]), H2D);
    hipMemcpy(Nknots, 
        table->nknots, n_coeffs*sizeof(table->nknots[0]), H2D);
    total_nknots = 0;
    for(index_t i=0; i<table->ndim; ++i) 
    {
        hipMemcpy(&(Knots[total_nknots]), 
            table->knots[i], table->nknots[i]*sizeof(table->knots[0]), H2D);
        total_nknots += table->nknots[i];
    }
    hipMemcpy(Coefficients, 
        table->coefficients, n_coeffs*sizeof(table->coefficients[0]), H2D);
    hipMemcpy(Naxes, 
        table->naxes, table->ndim*sizeof(table->naxes[0]), H2D);
    hipMemcpy(Strides, 
        table->strides, table->ndim*sizeof(table->strides[0]), H2D);

    // Bind the data
    hipMemcpy(&(Table->order), 
        &Order, sizeof(Table->order), H2D);
    hipMemcpy(&(Table->nknots), 
        &Nknots, sizeof(Table->order), H2D);
    hipMemcpy(&(Table->knots), 
        &Knots, sizeof(Table->order), H2D);
    hipMemcpy(&(Table->coefficients), 
        &Coefficients, sizeof(Table->order), H2D);
    hipMemcpy(&(Table->naxes), 
        &Naxes, sizeof(Table->order), H2D);
    hipMemcpy(&(Table->strides), 
        &Strides, sizeof(Table->order), H2D);
}


// void GPU_BSPLV::table_to_gpu(
//     splinetable * table)
// {
//     Splinetable * Table = nullptr;
//     hipMalloc(&Table, sizeof(*Table));
//     index_t * Order = nullptr;
//     value_t * Knots = nullptr, * Coefficients = nullptr;
//     long * Nknots = nullptr, * Naxes = nullptr;
//     unsigned long * Strides = nullptr;

//     // Allocate the arrays
//     hipMalloc(&Order, table->ndim*sizeof(*Order));
//     // Get the total number of knots
//     index_t total_nknots = 0;
//     for(index_t k=0; k<table->ndim; ++k) total_nknots += table->nknots[k];
//     hipMalloc(&Knots, total_nknots*sizeof(*Knots));
//     hipMalloc(&Nknots, table->ndim*sizeof(*Nknots));
//     // Extends?
//     // Periods?
//     // Get the total number of coefficients
//     index_t n_coeffs = 0;
//     for(index_t c=0; c<table->ndim; ++c) n_coeffs += table->naxes[c];
//     hipMalloc(&Coefficients, n_coeffs*sizeof(*Coefficients));
//     hipMalloc(&Naxes, table->ndim*sizeof(*Naxes));
//     hipMalloc(&Strides, table->ndim*sizeof(*Strides));

//     // Copy the data
//     hipMemcpy(Order, 
//         table->order, table->ndim*sizeof(table->order[0]), H2D);
//     hipMemcpy(Nknots, 
//         table->nknots, n_coeffs*sizeof(table->nknots[0]), H2D);
//     total_nknots = 0;
//     for(index_t i=0; i<table->ndim; ++i) 
//     {
//         hipMemcpy(&(Knots[total_nknots]), 
//             table->knots[i], table->nknots[i]*sizeof(table->knots[0]), H2D);
//         total_nknots += table->nknots[i];
//     }
//     hipMemcpy(Coefficients, 
//         table->coefficients, n_coeffs*sizeof(table->coefficients[0]), H2D);
//     hipMemcpy(Naxes, 
//         table->naxes, table->ndim*sizeof(table->naxes[0]), H2D);
//     hipMemcpy(Strides, 
//         table->strides, table->ndim*sizeof(table->strides[0]), H2D);

//     // Bind the data
//     hipMemcpy(&(Table->order), 
//         &Order, sizeof(Table->order), H2D);
//     hipMemcpy(&(Table->nknots), 
//         &Nknots, sizeof(Table->order), H2D);
//     hipMemcpy(&(Table->knots), 
//         &Knots, sizeof(Table->order), H2D);
//     hipMemcpy(&(Table->coefficients), 
//         &Coefficients, sizeof(Table->order), H2D);
//     hipMemcpy(&(Table->naxes), 
//         &Naxes, sizeof(Table->order), H2D);
//     hipMemcpy(&(Table->strides), 
//         &Strides, sizeof(Table->order), H2D);
//     return Table;
//     // // hipMalloc((void**)&Table, sizeof(Splinetable));
//     // // Table->order = nullptr;
    
//     // // Allocate the arrays
//     // hipMalloc(&(Table->order), table->ndim*sizeof(table->order[0]));
//     // // Get the total number of knots
//     // index_t total_nknots = 0;
//     // for(index_t k=0; k<table->ndim; ++k) total_nknots += table->nknots[k];
//     // hipMalloc(&(Table->knots), total_nknots*sizeof(table->knots[0]));
//     // hipMalloc(&(Table->nknots), table->ndim*sizeof(table->nknots[0]));
//     // // Extends?
//     // // Periods?
//     // // Get the total number of coefficients
//     // index_t n_coeffs = 0;
//     // for(index_t c=0; c<table->ndim; ++c) n_coeffs += table->naxes[c];
//     // hipMalloc(&(Table->coefficients), n_coeffs*sizeof(table->coefficients[0]));
//     // hipMalloc(&(Table->naxes), table->ndim*sizeof(table->naxes[0]));
//     // hipMalloc(&(Table->strides), table->ndim*sizeof(table->strides[0]));

//     // // Copy the arrays
//     // hipMemcpy(Table->order, 
//     //     table->order, table->ndim*sizeof(table->order[0]), H2D);
//     // total_nknots = 0;
//     // for(index_t i=0; i<table->ndim; ++i) 
//     // {
//     //     hipMemcpy(&(Table->knots[total_nknots]), 
//     //         table->knots[i], table->nknots[i]*sizeof(table->knots[0]), H2D);
//     //     total_nknots += table->nknots[i];
//     // }

//     // hipMemcpy(Table->coefficients, 
//     //     table->coefficients, n_coeffs*sizeof(table->coefficients[0]), H2D);
//     // hipMemcpy(Table->naxes, 
//     //     table->naxes, table->ndim*sizeof(table->naxes[0]), H2D);
//     // hipMemcpy(Table->strides, 
//     //     table->strides, table->ndim*sizeof(table->strides[0]), H2D);

//     // return Table;
// }

// void GPU_BSPLV::gpu_eval_splines(
//     Splinetable * Table, 
//     index_t ndim,
//     index_t n_evals,
//     value_t * y_array)
// {

// }

GPU_BSPLV::~GPU_BSPLV()
{
    hipFree(Order);
    hipFree(Knots);
    hipFree(Coefficients);
    hipFree(Naxes);
    hipFree(Strides);
    hipFree(Table);
}
